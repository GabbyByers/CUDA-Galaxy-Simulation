#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"

__global__ void gpuStarProjection(Star* device_stars, int num_stars, CanvasCamera* device_canvas_camera) {
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= num_stars) {
        return;
    }

    Star& star = device_stars[index];
    star.projection(device_canvas_camera);
}

// Canvas Camera
CanvasCamera* allocateDeviceCanvasCamera() {
    CanvasCamera* device_canvas_camera = nullptr;
    hipMalloc((void**)&device_canvas_camera, sizeof(CanvasCamera));
    return device_canvas_camera;
}

void sendCanvasCameraToDevice(CanvasCamera* device_canvas_camera, CanvasCamera* host_canvas_camera) {
    hipMemcpy(device_canvas_camera, host_canvas_camera, sizeof(CanvasCamera), hipMemcpyHostToDevice);
}

void freeDeviceCanvasCamera(CanvasCamera* device_canvas_camera) {
    hipFree(device_canvas_camera);
}

// Stars
Star* allocateDeviceStars(Star* host_stars, int num_stars) {
    hipSetDevice(0);

    Star* device_stars = nullptr;
    hipMalloc((void**)&host_stars, num_stars * sizeof(Star));

    hipMemcpy(device_stars, host_stars, num_stars * sizeof(Star), hipMemcpyHostToDevice);

    return device_stars;
}

void deviceStarProjection(Star* device_stars, int num_stars, CanvasCamera* device_canvas_camera) {
    unsigned int NUM_THREADS = 512;
    unsigned int NUM_BLOCKS = (num_stars + NUM_THREADS - 1) / NUM_THREADS;
    gpuStarProjection <<<NUM_BLOCKS, NUM_THREADS>>> (device_stars, num_stars, device_canvas_camera);
    hipDeviceSynchronize();

    
}

void copyDeviceStarsToHostStars(Star* host_stars, Star* device_stars, int num_stars) {
    hipMemcpy(host_stars, device_stars, num_stars * sizeof(Star), hipMemcpyDeviceToHost);
}

void freeDeviceStars(Star* device_stars) {
    hipFree(device_stars);
}