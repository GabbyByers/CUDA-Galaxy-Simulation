#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"

__global__ void gpuStarProjection(Star* stars, int num_stars) {
    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= num_stars) {
        return;
    }
}

Star* initializeDeviceStars(Star* host_stars, int num_stars) {
    hipSetDevice(0);

    Star* device_stars = nullptr;
    hipMalloc((void**)&host_stars, num_stars * sizeof(Star));

    hipMemcpy(device_stars, host_stars, num_stars * sizeof(Star), hipMemcpyHostToDevice);

    return device_stars;
}

void deviceStarProjection(Star* host_stars, Star* device_stars, int num_stars) {
    unsigned int NUM_THREADS = 512;
    unsigned int NUM_BLOCKS = (num_stars + NUM_THREADS - 1) / NUM_THREADS;
    gpuStarProjection <<<NUM_BLOCKS, NUM_THREADS>>> (device_stars, num_stars);
    hipDeviceSynchronize();

    hipMemcpy(host_stars, device_stars, num_stars * sizeof(Star), hipMemcpyDeviceToHost);
}

void freeDeviceStars(Star* device_stars) {
    hipFree(device_stars);
}